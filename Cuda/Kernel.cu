#include "hip/hip_runtime.h"
#include "Kernel.cuh"

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void handleMovementKernel(unsigned numParticles, float deltaTime, float* particlePos, float* particleVel, bool GRAVITY) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < numParticles) {
        if (GRAVITY) {
            particleVel[tid * 2 + 1] -= 10.0f * deltaTime;
        }
        particlePos[tid * 2] += particleVel[tid * 2] * deltaTime;
        particlePos[tid * 2 + 1] += particleVel[tid * 2 + 1] * deltaTime;
    }
}

void launchMovementKernel(unsigned numParticles, float deltaTime, float* h_particlePos, float* h_particleVel, bool GRAVITY) {

	unsigned numBlocks = (numParticles + blockSize - 1) / blockSize; 

	// Kernel launch preprocessing
    float* d_particlePos;
    float* d_particleVel;

    // Memory allocation: Device
    hipMalloc(&d_particlePos, numParticles * 2 * sizeof(float));
    cudaCheckErrors("Malloc failure: Particle Positions");
    hipMalloc(&d_particleVel, numParticles * 2 * sizeof(float));
    cudaCheckErrors("Malloc failure: Particle Velocities");

    // Memory copy: Host to device
    hipMemcpy(d_particlePos, h_particlePos, numParticles * 2 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("Memcpy failure: Particle Positions host to device");
    hipMemcpy(d_particleVel, h_particleVel, numParticles * 2 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("Memcpy failure: Particle Velocities host to device");

    // Kernel Launch
	handleMovementKernel <<< numBlocks, blockSize >>> (numParticles, deltaTime, d_particlePos, d_particleVel, GRAVITY);
    hipDeviceSynchronize();
    cudaCheckErrors("Kernel Launch: Calculate position kernel");

    // Memory copy: Device to host
    hipMemcpy(h_particlePos, d_particlePos, numParticles * 2 * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("Memcpy failure: Particle Positions device to host");
    hipMemcpy(h_particleVel, d_particleVel, numParticles * 2 * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("Memcpy failure: Particle Velocities device to host");

    hipFree(d_particlePos);
    cudaCheckErrors("Free failure: Particle Positions");
    hipFree(d_particleVel);
    cudaCheckErrors("Free failure: Particle Velocities");


}