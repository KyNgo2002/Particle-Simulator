#include "hip/hip_runtime.h"
#include "Kernel.cuh"

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

__global__ void handleMovementKernel(unsigned numParticles, float deltaTime, float* particlePos, float* particleVel, bool GRAVITY) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < numParticles) {
        if (GRAVITY) {
            particleVel[tid * 2 + 1] -= 10.0f * deltaTime;
        }
        particlePos[tid * 2] += particleVel[tid * 2] * deltaTime;
        particlePos[tid * 2 + 1] += particleVel[tid * 2 + 1] * deltaTime;
    }
}

void launchMovementKernel(CudaHelper& cudaHelper, float deltaTime) {

	unsigned numBlocks = (cudaHelper.m_numParticles + blockSize - 1) / blockSize; 

    // Memory copy: Host to device
    hipMemcpy(cudaHelper.d_particlePos, cudaHelper.h_particlePos, cudaHelper.m_numParticles * 2 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("Memcpy failure: Particle Positions host to device");
    hipMemcpy(cudaHelper.d_particleVel, cudaHelper.h_particleVel, cudaHelper.m_numParticles * 2 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("Memcpy failure: Particle Velocities host to device");

    // Kernel Launch
	handleMovementKernel <<< numBlocks, blockSize >>> (cudaHelper.m_numParticles, deltaTime, cudaHelper.d_particlePos, cudaHelper.d_particleVel, cudaHelper.m_GRAVITY);
    hipDeviceSynchronize();
    cudaCheckErrors("Kernel Launch: Calculate position kernel");

    // Memory copy: Device to host
    hipMemcpy(cudaHelper.h_particlePos, cudaHelper.d_particlePos, cudaHelper.m_numParticles * 2 * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("Memcpy failure: Particle Positions device to host");
    hipMemcpy(cudaHelper.h_particleVel, cudaHelper.d_particleVel, cudaHelper.m_numParticles * 2 * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("Memcpy failure: Particle Velocities device to host");

}